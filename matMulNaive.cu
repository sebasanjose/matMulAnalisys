#include <stdio.h>
#include <hip/hip_runtime.h>

#define WIDTH 2048  // Large matrix size

// Naïve matrix multiplication (No tiling, inefficient)
__global__ void matMulNaive(float *A, float *B, float *C, int Width) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < Width && Col < Width) {
        float Pvalue = 0.0;
        for (int k = 0; k < Width; k++) {
            Pvalue += A[Row * Width + k] * B[k * Width + Col];
        }
        C[Row * Width + Col] = Pvalue;
    }
}

int main() {
    size_t size = WIDTH * WIDTH * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize matrices
    for (int i = 0; i < WIDTH * WIDTH; i++) {
        h_A[i] = static_cast<float>(rand() % 10);
        h_B[i] = static_cast<float>(rand() % 10);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (WIDTH + blockSize.y - 1) / blockSize.y);

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Execute the naive kernel
    matMulNaive<<<gridSize, blockSize>>>(d_A, d_B, d_C, WIDTH);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print input matrices and output matrix
    printf("Matrix A:\n");
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%0.2f ", h_A[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("\nMatrix B:\n");
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%0.2f ", h_B[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("\nMatrix C (Result):\n");
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%0.2f ", h_C[i * WIDTH + j]);
        }
        printf("\n");
    }

    printf("\nElapsed Time: %f ms\n", elapsedTime);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}