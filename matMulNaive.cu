#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define WIDTH 2048  // Large matrix size
#define EPSILON 1e-4

// Naïve matrix multiplication (No tiling, inefficient)
__global__ void matMulNaive(float *A, float *B, float *C, int Width) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < Width && Col < Width) {
        float Pvalue = 0.0;
        for (int k = 0; k < Width; k++) {
            Pvalue += A[Row * Width + k] * B[k * Width + Col];
        }
        C[Row * Width + Col] = Pvalue;
    }
}

// Function to verify the results
bool verifyResult(float *A, float *B, float *C, int Width) {
    for (int i = 0; i < Width; i++) {
        for (int j = 0; j < Width; j++) {
            float expected = 0.0;
            for (int k = 0; k < Width; k++) {
                expected += A[i * Width + k] * B[k * Width + j];
            }
            if (fabs(C[i * Width + j] - expected) > EPSILON) {
                printf("Mismatch at (%d, %d): expected %0.4f, got %0.4f\n", i, j, expected, C[i * Width + j]);
                return false;
            }
        }
    }
    return true;
}

int main() {
    size_t size = WIDTH * WIDTH * sizeof(float);

    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize matrices
    for (int i = 0; i < WIDTH * WIDTH; i++) {
        h_A[i] = static_cast<float>(rand() % 10);
        h_B[i] = static_cast<float>(rand() % 10);
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (WIDTH + blockSize.y - 1) / blockSize.y);

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Execute the naive kernel
    matMulNaive<<<gridSize, blockSize>>>(d_A, d_B, d_C, WIDTH);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    if (verifyResult(h_A, h_B, h_C, WIDTH)) {
        printf("Matrix multiplication is correct!\n");
    } else {
        printf("Matrix multiplication verification failed!\n");
    }

    // Print elapsed time
    printf("\nElapsed Time: %f ms\n", elapsedTime);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}